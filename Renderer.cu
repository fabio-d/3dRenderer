#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cmath>

#include <GL/gl.h>
#include <hip/hip_runtime.h>

#include "Point.h"
#include "Renderer.h"
#include "Triangle.h"

static __global__ void clearBuffersK(char *colorbuffer, float *depthbuffer,
	int npixels, char r, char g, char b)
{
	const int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < npixels)
	{
		char *pixel = colorbuffer + i * 3;
		*pixel++ = r;
		*pixel++ = g;
		*pixel = b;

		depthbuffer[i] = INFINITY;
	}
}

static __global__ void drawLineK(char *colorbuffer, float *depthbuffer, float subdivisions,
	float dx, float dy, float dz, float x1, float y1, float z1,
	int w, char r, char g, char b)
{
	const float pos = (threadIdx.x + blockIdx.x * blockDim.x) / (subdivisions - 1);

	if (pos <= 1)
	{
		const int x = roundf(x1 + dx * pos);
		const int y = roundf(y1 + dy * pos);
		const float z = z1 + dz * pos;

		const int i = x + y * w;

		if (depthbuffer[i] > z)
		{
			char *pixel = colorbuffer + i * 3;
			*pixel++ = r;
			*pixel++ = g;
			*pixel = b;

			depthbuffer[i] = z;
		}
	}
}

static __global__ void drawTriangleK(char *colorbuffer, float *depthbuffer,
	int x0, int y0, float z0, int x1, int y1, float z1, int x2, int y2, float z2,
	int min_x, int min_y, int w, char r, char g, char b)
{
	const int x = min_x + threadIdx.x + blockIdx.x * blockDim.x;
	const int y = min_y + threadIdx.y + blockIdx.y * blockDim.y;

	// Controlla che il punto sia interno ai tre lati del triangolo
	const float prodscal01 = (x1-x0)*(y-y0) - (y1-y0)*(x-x0);
	const float prodscal12 = (x2-x1)*(y-y1) - (y2-y1)*(x-x1);
	const float prodscal20 = (x0-x2)*(y-y2) - (y0-y2)*(x-x2);
	if ( (prodscal01 <= 0 && prodscal12 <= 0 && prodscal20 <= 0) ||
		(prodscal01 >= 0 && prodscal12 >= 0 && prodscal20 >= 0) )
	{
		const int i = x + y * w;
		const float z = (((x0-x)*y1+(x-x1)*y0+(x1-x0)*y)*z2+((x-x0)*y2+(x2-x)*y0+(x0-x2)*y)*z1+((x1-x)*y2+(x-x2)*y1+(x2-x1)*y)*z0)/((x1-x0)*y2+(x0-x2)*y1+(x2-x1)*y0);

		if (depthbuffer[i] > z)
		{
			char *pixel = colorbuffer + i * 3;
			*pixel++ = r;
			*pixel++ = g;
			*pixel = b;

			depthbuffer[i] = z;
		}
	}
}

Renderer::Renderer()
: buffer_w(0), buffer_h(0), colorbuffer(0), drawMode(Solid),
  stroke_r(0), stroke_g(0), stroke_b(0),
  fill_r(0), fill_g(0), fill_b(0)
{
}

void Renderer::resize(int w, int h)
{
	fprintf(stderr, "Buffer ridimensionato a %dx%d\n", w, h);

	if (colorbuffer != 0)
	{
		hipFree(colorbuffer);
		hipFree(depthbuffer);
	}

	hipError_t err = hipMalloc(&colorbuffer, w * h * 3);
	if (err != hipSuccess)
	{
		fprintf(stderr, "hipMalloc (colorbuffer) fallito: %d - %s\n", err, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc(&depthbuffer, w * h * sizeof(float));
	if (err != hipSuccess)
	{
		fprintf(stderr, "hipMalloc (depthbuffer) fallito: %d - %s\n", err, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	buffer_w = w;
	buffer_h = h;
}

void Renderer::copyToScreen()
{
	const int buffsize = buffer_w * buffer_h * 3;
	char * temp = (char*)malloc(buffsize);
	if (temp == 0)
	{
		fprintf(stderr, "memoria esaurita in copyToScreen\n");
		exit(EXIT_FAILURE);
	}

	hipError_t err = hipMemcpy(temp, colorbuffer, buffsize, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy fallito: %d - %s\n", err, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	glRasterPos2i(-1, -1);
	glDrawPixels(buffer_w, buffer_h, GL_RGB, GL_UNSIGNED_BYTE, temp);

	free(temp);
}

void Renderer::resetMatrixStack()
{
	mtxStack.clear();
	currMatrix = Matrix::scale(buffer_h / 4.f, buffer_h / 4.f, 1) * Matrix::translate(buffer_w / 2.f, buffer_h / 2.f, 0);
}

void Renderer::pushMatrix()
{
	mtxStack.push_back(currMatrix);
}

void Renderer::popMatrix()
{
	currMatrix = mtxStack.back();
	mtxStack.pop_back();
}

void Renderer::multMatrix(const Matrix &mtx)
{
	currMatrix = mtx * currMatrix;
}

void Renderer::drawTriangle(const Triangle &t)
{
	if (drawMode == Solid || drawMode == WireframeHiddenLineRemoval)
	{
		drawTriangleNoTransform(
			Point4(t.vA) * currMatrix,
			Point4(t.vB) * currMatrix,
			Point4(t.vC) * currMatrix,
			fill_r, fill_g, fill_b);
	}

	if (drawMode == Wireframe || drawMode == WireframeHiddenLineRemoval)
	{
		drawLine(t.vA, t.vB);
		drawLine(t.vB, t.vC);
		drawLine(t.vC, t.vA);
	}
}

void Renderer::setDrawMode(DrawMode newMode)
{
	drawMode = newMode;
}

Renderer::DrawMode Renderer::getDrawMode() const
{
	return drawMode;
}

void Renderer::setStrokeColor(char r, char g, char b)
{
	stroke_r = r;
	stroke_g = g;
	stroke_b = b;
}

void Renderer::setFillColor(char r, char g, char b)
{
	fill_r = r;
	fill_g = g;
	fill_b = b;
}

void Renderer::drawLine(const Point3 &p1, const Point3 &p2)
{
	drawLineNoTransform(Point4(p1) * currMatrix, Point4(p2) * currMatrix);
}

Renderer::~Renderer()
{
	if (colorbuffer != 0)
	{
		hipFree(colorbuffer);
		hipFree(depthbuffer);
	}
}

void Renderer::clearScreen()
{
	const int npixels = buffer_w * buffer_h;

	const int blockSize = 32;
	const int numBlocks = (npixels + blockSize - 1) / blockSize;

	clearBuffersK<<<numBlocks, blockSize>>>(colorbuffer, depthbuffer,
		npixels, fill_r, fill_g, fill_b);

	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess)
	{
		fprintf(stderr, "clearBuffersK fallito: %d - %s\n", err, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void Renderer::drawLineNoTransform(const Point4 &p1, const Point4 &p2)
{
	const int x1 = roundf(p1.x / p1.t);
	const int y1 = roundf(p1.y / p1.t);
	const int x2 = roundf(p2.x / p2.t);
	const int y2 = roundf(p2.y / p2.t);

	const int subdivisions = 2 * ceil(sqrt(pow(x2-x1, 2) + pow(y2-y1, 2)));
	const float dx = (x2 - x1);
	const float dy = (y2 - y1);
	const float dz = (p2.z - p1.z);

	const int blockSize = 32;
	const int numBlocks = (subdivisions + blockSize - 1) / blockSize;

	drawLineK<<<numBlocks, blockSize>>>(colorbuffer, depthbuffer, subdivisions,
		dx, dy, dz, x1, y1, p1.z, buffer_w,
		stroke_r, stroke_g, stroke_b);

	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess)
	{
		fprintf(stderr, "drawLineK fallito: %d - %s\n", err, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void Renderer::drawTriangleNoTransform(const Point4 &p0, const Point4 &p1,
	const Point4 &p2, char r, char g, char b)
{
	const int x0 = roundf(p0.x / p0.t);
	const int y0 = roundf(p0.y / p0.t);
	const float z0 = p0.z;
	const int x1 = roundf(p1.x / p1.t);
	const int y1 = roundf(p1.y / p1.t);
	const float z1 = p1.z;
	const int x2 = roundf(p2.x / p2.t);
	const int y2 = roundf(p2.y / p2.t);
	const float z2 = p2.z;

	// Calcola bounding box
	const float min_x = min(min(x0, x1), x2);
	const float max_x = max(max(x0, x1), x2);
	const float min_y = min(min(y0, y1), y2);
	const float max_y = max(max(y0, y1), y2);

	dim3 blockSize;
	blockSize.x = 32;
	blockSize.y = 32;

	dim3 numBlocks;
	numBlocks.x = ( (max_x - min_x + 1) + blockSize.x - 1 ) / blockSize.x;
	numBlocks.y = ( (max_y - min_y + 1) + blockSize.y - 1 ) / blockSize.y;

	drawTriangleK<<<numBlocks, blockSize>>>(colorbuffer, depthbuffer,
		x0, y0, z0, x1, y1, z1, x2, y2, z2,
		min_x, min_y, buffer_w, r, g, b);
}
