#include "hip/hip_runtime.h"
#include "CudaRenderer.h"

#include <cstdio>
#include <cstdlib>
#include <cmath>

#include <GL/gl.h>
#include <hip/hip_runtime.h>

#include "Point.h"
#include "Triangle.h"

static __global__ void clearBuffersK(char *colorbuffer, float *depthbuffer,
	int npixels, char r, char g, char b)
{
	const int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < npixels)
	{
		char *pixel = colorbuffer + i * 3;
		*pixel++ = r;
		*pixel++ = g;
		*pixel = b;

		depthbuffer[i] = INFINITY;
	}
}

static __global__ void drawLineK(char *colorbuffer, float *depthbuffer, float subdivisions,
	float dx, float dy, float dz, float x1, float y1, float z1,
	int w, int h, char r, char g, char b)
{
	const float pos = (threadIdx.x + blockIdx.x * blockDim.x) / (subdivisions - 1);

	if (pos <= 1)
	{
		const int x = roundf(x1 + dx * pos);
		const int y = roundf(y1 + dy * pos);

		if (x < 0 || x > w || y < 0 || y > h)
			return;

		const float z = z1 + dz * pos;

		const int i = x + y * w;

		if (depthbuffer[i] > z)
		{
			char *pixel = colorbuffer + i * 3;
			*pixel++ = r;
			*pixel++ = g;
			*pixel = b;

			depthbuffer[i] = z;
		}
	}
}

static __global__ void drawTriangleK(char *colorbuffer, float *depthbuffer,
	int x0, int y0, float z0, int x1, int y1, float z1, int x2, int y2, float z2,
	int min_x, int min_y, int w, int h, char r, char g, char b)
{
	const int x = min_x + threadIdx.x + blockIdx.x * blockDim.x;
	const int y = min_y + threadIdx.y + blockIdx.y * blockDim.y;

	if (x < 0 || x > w || y < 0 || y > h)
		return;

	// Controlla che il punto sia interno ai tre lati del triangolo
	const float prodscal01 = (x1-x0)*(y-y0) - (y1-y0)*(x-x0);
	const float prodscal12 = (x2-x1)*(y-y1) - (y2-y1)*(x-x1);
	const float prodscal20 = (x0-x2)*(y-y2) - (y0-y2)*(x-x2);
	if ( (prodscal01 <= 0 && prodscal12 <= 0 && prodscal20 <= 0) ||
		(prodscal01 >= 0 && prodscal12 >= 0 && prodscal20 >= 0) )
	{
		const int i = x + y * w;
		const float z = (((x0-x)*y1+(x-x1)*y0+(x1-x0)*y)*z2+((x-x0)*y2+(x2-x)*y0+(x0-x2)*y)*z1+((x1-x)*y2+(x-x2)*y1+(x2-x1)*y)*z0)/((x1-x0)*y2+(x0-x2)*y1+(x2-x1)*y0);

		if (depthbuffer[i] > z)
		{
			char *pixel = colorbuffer + i * 3;
			*pixel++ = r;
			*pixel++ = g;
			*pixel = b;

			depthbuffer[i] = z;
		}
	}
}

CudaRenderer::CudaRenderer()
: buffer_w(0), buffer_h(0), colorbuffer(0)
{
}

void CudaRenderer::setGeometry(int w, int h)
{
	fprintf(stderr, "Buffer ridimensionato a %dx%d\n", w, h);

	if (colorbuffer != 0)
	{
		hipFree(colorbuffer);
		hipFree(depthbuffer);
	}

	hipError_t err = hipMalloc(&colorbuffer, w * h * 3);
	if (err != hipSuccess)
	{
		fprintf(stderr, "hipMalloc (colorbuffer) fallito: %d - %s\n", err, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc(&depthbuffer, w * h * sizeof(float));
	if (err != hipSuccess)
	{
		fprintf(stderr, "hipMalloc (depthbuffer) fallito: %d - %s\n", err, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	buffer_w = w;
	buffer_h = h;

	normMatrix = Matrix::translate(1, 1, 0) * Matrix::scale(buffer_w / 2.f, buffer_h / 2.f, 1);
}

void CudaRenderer::flushToFramebuffer()
{
	const int buffsize = buffer_w * buffer_h * 3;
	char * temp = (char*)malloc(buffsize);
	if (temp == 0)
	{
		fprintf(stderr, "memoria esaurita in flushToFramebuffer\n");
		exit(EXIT_FAILURE);
	}

	hipError_t err = hipMemcpy(temp, colorbuffer, buffsize, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy fallito: %d - %s\n", err, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	glRasterPos2i(-1, -1);
	glDrawPixels(buffer_w, buffer_h, GL_RGB, GL_UNSIGNED_BYTE, temp);

	free(temp);
}

void CudaRenderer::clearBuffers()
{
	char r, g, b;
	color(&r, &g, &b);

	const int npixels = buffer_w * buffer_h;

	const int blockSize = 32;
	const int numBlocks = (npixels + blockSize - 1) / blockSize;

	clearBuffersK<<<numBlocks, blockSize>>>(colorbuffer, depthbuffer,
		npixels, r, g, b);

	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess)
	{
		fprintf(stderr, "clearBuffersK fallito: %d - %s\n", err, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void CudaRenderer::drawLine(const Point3 &p1, const Point3 &p2)
{
	const Matrix mtx = matrix() * normMatrix;

	drawLineNoTransform(Point4(p1) * mtx, Point4(p2) * mtx);
}

void CudaRenderer::drawTriangle(const Triangle &t)
{
	const Matrix mtx = matrix() * normMatrix;

	const Point4 tA = Point4(t.vA) * mtx;
	const Point4 tB = Point4(t.vB) * mtx;
	const Point4 tC = Point4(t.vC) * mtx;

	drawTriangleNoTransform(tA, tB, tC);
}

CudaRenderer::~CudaRenderer()
{
	if (colorbuffer != 0)
	{
		hipFree(colorbuffer);
		hipFree(depthbuffer);
	}
}

void CudaRenderer::drawLineNoTransform(const Point4 &p1, const Point4 &p2)
{
	char r, g, b;
	color(&r, &g, &b);

	const int x1 = roundf(p1.x / p1.t);
	const int y1 = roundf(p1.y / p1.t);
	const int x2 = roundf(p2.x / p2.t);
	const int y2 = roundf(p2.y / p2.t);

	const int subdivisions = 2 * ceil(sqrt(pow(x2-x1, 2) + pow(y2-y1, 2)));
	const float dx = (x2 - x1);
	const float dy = (y2 - y1);
	const float dz = (p2.z - p1.z);

	const int blockSize = 32;
	const int numBlocks = (subdivisions + blockSize - 1) / blockSize;

	drawLineK<<<numBlocks, blockSize>>>(colorbuffer, depthbuffer, subdivisions,
		dx, dy, dz, x1, y1, p1.z, buffer_w, buffer_h,
		r, g, b);

	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess)
	{
		fprintf(stderr, "drawLineK fallito: %d - %s\n", err, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void CudaRenderer::drawTriangleNoTransform(const Point4 &p0, const Point4 &p1,
	const Point4 &p2)
{
	char r, g, b;
	color(&r, &g, &b);

	const int x0 = roundf(p0.x / p0.t);
	const int y0 = roundf(p0.y / p0.t);
	const float z0 = p0.z;
	const int x1 = roundf(p1.x / p1.t);
	const int y1 = roundf(p1.y / p1.t);
	const float z1 = p1.z;
	const int x2 = roundf(p2.x / p2.t);
	const int y2 = roundf(p2.y / p2.t);
	const float z2 = p2.z;

	// Calcola bounding box
	const float min_x = min(min(x0, x1), x2);
	const float max_x = max(max(x0, x1), x2);
	const float min_y = min(min(y0, y1), y2);
	const float max_y = max(max(y0, y1), y2);

	dim3 blockSize;
	blockSize.x = 16;
	blockSize.y = 16;

	dim3 numBlocks;
	numBlocks.x = ( (max_x - min_x + 1) + blockSize.x - 1 ) / blockSize.x;
	numBlocks.y = ( (max_y - min_y + 1) + blockSize.y - 1 ) / blockSize.y;

	drawTriangleK<<<numBlocks, blockSize>>>(colorbuffer, depthbuffer,
		x0, y0, z0, x1, y1, z1, x2, y2, z2,
		min_x, min_y, buffer_w, buffer_h, r, g, b);
}
